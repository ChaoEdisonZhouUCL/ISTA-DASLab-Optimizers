#include "hip/hip_runtime.h"
#include "../utils.h"

__global__ void symm_block_quant_inv_kernel(LL d, LL q_block_size, uint8_t *xq, bfloat16 *ranges, bfloat16 *x) {
	/*
		This kernel computes x += Q_inv(xq, range) for 4 bits (implements point 1 from PhD notebook page 55)
		Here, x is the output buffer and will already contain the dense gradient
		The output buffer x has d components and xq has d/2 components because one uint8_t stores two 4-bit values
		In contrast to "globally" kernel, this kernel works with a single block
		Make sure block_size is always divisible by 2

		We have to read:
		- q_block_size values from x
		- one value from ranges
		- q_block_size / 2 values from xq
	*/
	bfloat162 *x2 = reinterpret_cast<bfloat162*>(x); // we will read two values from x at once

	const LL B = gridDim.x; // number of blocks
	const LL Bid = blockIdx.x; // block id
	const LL T = blockDim.x; // number of threads
	const LL Tid = threadIdx.x; // thread id

	LL half_d = (d >> 1);
	LL half_q_block_size = (q_block_size >> 1); // block size in xq
	LL half_start_index = Bid * half_q_block_size; // start index in vector x
	LL half_end_index = min(half_start_index + half_q_block_size, half_d); // end index in vector x
	float r = __bfloat162float(ranges[Bid]);
    float S = 2.0f * r / 15.0f; // 15 = 16 - 1 = 2^4 - 1

	bfloat162 vx2; // the value that will store x2[index]
	uint8_t vq; // the value that will store xq[index]
	uint8_t msb; // the MSB of a xq component
	uint8_t lsb; // the LSB of a xq component

	for(LL half_index = half_start_index + Tid; half_index < half_end_index; half_index += T) {
        vx2 = x2[half_index];
        vq = xq[half_index];

		msb = (vq & 0xF0) >> 4;
		lsb = (vq & 0x0F);

        // += operation happens here
// 		vx2.x += __float2bfloat16((msb + 8.0f) * S - r);  // xq in [-2^(b-1), 2^(b-1)-1]
// 		vx2.y += __float2bfloat16((lsb + 8.0f) * S - r);
		vx2.x += __float2bfloat16(msb * S - r);  // xq in [0, 2^b-1]
		vx2.y += __float2bfloat16(lsb * S - r);
		x2[half_index] = vx2;
		// printf"[Qinv] Bid=%ld, Tid=%ld, half_index=%ld, r=%f, S=%f, vx2.x=%f, vx2.y=%f, msb=%d, lsb=%d\n",
// 		    Bid, Tid, half_index, r, S, __bfloat162float(vx2.x), __bfloat162float(vx2.y), msb, lsb);
	}
	if((d & 1) && (Bid == B-1) && (Tid == T-1)) {
		bfloat16 vx = x[d - 1];
		vq = xq[half_d];
		msb = (vq & 0xF0) >> 4;
// 		vx += __float2bfloat16((msb + 8.0f) * S - r);  // xq in [-2^(b-1), 2^(b-1)-1]
		vx += __float2bfloat16(msb * S - r);  // xq in [0, 2^b-1]
		x[d - 1] = vx;
		// printf"[Qinv] Bid=%ld, Tid=%ld, last, r=%f, S=%f, vx=%f, msb=%d, lsb=%d\n",
// 		    Bid, Tid, r, S, __bfloat162float(vx), msb, lsb);
	}
}
void symm_block_quant_inv_cuda(LL d, LL q_block_size, torch::Tensor xq, torch::Tensor ranges, torch::Tensor x) {
    LL blocks = 1 + (LL)(d / q_block_size);
    symm_block_quant_inv_kernel<<<blocks, 1024>>>(d,
                                                    q_block_size,
                                                    (uint8_t*) xq.data_ptr(),
                                                    (bfloat16*) ranges.data_ptr(),
                                                    (bfloat16*) x.data_ptr());
    // error checks
	gpuErrorCheck(hipGetLastError());
	gpuErrorCheck(hipPeekAtLastError());
// 	gpuErrorCheck(hipDeviceSynchronize());
}